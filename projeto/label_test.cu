#include <iostream>
#include <string>
#include <sstream>
#include <hip/hip_runtime.h>
#include "ppm.h"
#include "labeling_cuda.cu"


#define START_TIME hipEventRecord(start,0)
#define STOP_TIME  hipEventRecord(stop,0 ); \
                   hipEventSynchronize(stop); \
                   hipEventElapsedTime( &et, start, stop )


int main(int argc, char* argv[]) {
	hipEvent_t start,stop;
	hipEventCreate( &start );
	hipEventCreate( &stop );
	float et;
	int *label;

	for(int i=0; i<9; i++){
		std::stringstream fname;
		fname << "imagens/teste" << i << ".ppm";
		ppm image(fname.str());

		int w = image.width;
		int h = image.height;

		label = (int*)malloc(w*h*sizeof(int));

		START_TIME;
		CCL(image.r.data(), w, h, label);
		STOP_TIME;

		std::cout << et << "  " ;

		free(label);
	}
	std::cout << std::endl;

	return 0;
}
